#include "CDeviceHitList_alignmentHits.cuh"

#include "common.hpp"
#include "krnlAlignment.cuh"
#include "CTest.cuh"

/******************************* private ************************************/
#include <thrust/sort.h>

void sortBeforeAlignBackWard(
		const int q_begin,
		const thrust::device_vector<int>& qLengthArray,
		thrust::device_vector<int>& targetIDArray,
		thrust::device_vector<int>& queryIDArray,
		thrust::device_vector<int>& targetIndexArray,
		thrust::device_vector<int>& queryIndexArray,
		thrust::device_vector<int>& tHitLengthArray,
		thrust::device_vector<int>& qHitLengthArray,
		thrust::device_vector<int>& matchNumArray,
		thrust::device_vector<int>& scoreArray) {
	using namespace thrust;

	device_vector<int> alignmentSize(targetIDArray.size());
	thrust::transform(
			make_zip_iterator(
					make_tuple(
							make_permutation_iterator(
									qLengthArray.begin() - q_begin,
									queryIDArray.begin()
							),
							queryIndexArray.begin(),
							qHitLengthArray.begin()
					)
			),
			make_zip_iterator(
					make_tuple(
							make_permutation_iterator(
									qLengthArray.begin() - q_begin,
									queryIDArray.end()
							),
							queryIndexArray.end(),
							qHitLengthArray.end()
					)
			),
			alignmentSize.begin(),
			make_alignmentSizeBackward()
	);

	sort(
			make_zip_iterator(
					make_tuple(
							targetIDArray   .begin(),
							queryIDArray    .begin(),
							targetIndexArray.begin(),
							queryIndexArray .begin(),
							tHitLengthArray .begin(),
							qHitLengthArray .begin(),
							matchNumArray   .begin(),
							scoreArray      .begin(),
							alignmentSize   .begin()
					)
			),
			make_zip_iterator(
					make_tuple(
							targetIDArray   .end(),
							queryIDArray    .end(),
							targetIndexArray.end(),
							queryIndexArray .end(),
							tHitLengthArray .end(),
							qHitLengthArray .end(),
							matchNumArray   .end(),
							scoreArray      .end(),
							alignmentSize   .end()
					)
			),
			alignLengthBackward()
	);
}

void sortBeforeAlignForward(
		thrust::device_vector<int>& targetIDArray,
		thrust::device_vector<int>& queryIDArray,
		thrust::device_vector<int>& targetIndexArray,
		thrust::device_vector<int>& queryIndexArray,
		thrust::device_vector<int>& tHitLengthArray,
		thrust::device_vector<int>& qHitLengthArray,
		thrust::device_vector<int>& matchNumArray,
		thrust::device_vector<int>& scoreArray) {
	using namespace thrust;

	sort(
			make_zip_iterator(
					make_tuple(
							targetIDArray   .begin(),
							queryIDArray    .begin(),
							targetIndexArray.begin(),
							queryIndexArray .begin(),
							tHitLengthArray .begin(),
							qHitLengthArray .begin(),
							matchNumArray   .begin(),
							scoreArray      .begin()
					)
			),
			make_zip_iterator(
					make_tuple(
							targetIDArray   .end(),
							queryIDArray    .end(),
							targetIndexArray.end(),
							queryIndexArray .end(),
							tHitLengthArray .end(),
							qHitLengthArray .end(),
							matchNumArray   .end(),
							scoreArray      .end()
					)
			),
			alignLengthForward()
	);
}

/******************************** public ************************************/

void alignmentHits(
		const CHostSetting& s,
		const CDeviceHashTable& h,
		const CDeviceSeqList_query& q,
		const int t_begin,
		const int q_begin,
		thrust::device_vector<int>& targetIDArray,
		thrust::device_vector<int>& targetIndexArray,
		thrust::device_vector<int>& queryIDArray,
		thrust::device_vector<int>& queryIndexArray,
		thrust::device_vector<int>& tHitLengthArray,
		thrust::device_vector<int>& qHitLengthArray,
		thrust::device_vector<int>& matchNumArray,
		thrust::device_vector<int>& scoreArray) {
        using namespace thrust;
        #ifdef TIME_ATTACK
                float elapsed_time_ms=0.0f;
                hipEvent_t start, stop;
                hipEventCreate( &start );
                hipEventCreate( &stop  );
                hipEventRecord( start, 0 );
                std::cout << "  ...allignment seeds";
        #endif /* TIME_ATTACK */
        const int initBlockDim_x  = 256;
	const int alignBlockDim_x = 32;
	const int hitNum = targetIDArray.size();
        const int allowableGap = s.getAllowableGap();
        const int tempNodeWidth = 1 + 2 * (allowableGap + MARGIN);
        const int tempNodeArraySize = hitNum * tempNodeWidth;
        device_vector<int> tempNodeArray_score     (tempNodeArraySize);
        device_vector<int> tempNodeArray_vertical  (tempNodeArraySize);
        device_vector<int> tempNodeArray_horizontal(tempNodeArraySize);
        device_vector<int> tempNodeArray_matchNum  (tempNodeArraySize);
        const int  initBlockNum = (tempNodeArraySize / initBlockDim_x) + 1;
        const dim3 initTempNodeBlock(65535, (initBlockNum/65535)+1, 1);
        const int  alignBlockNum = (hitNum/alignBlockDim_x)+1;
        const dim3 alignNodeBlock(65535, (alignBlockNum/65535)+1, 1);
	if(s.getFlgLocal()) {
		sortBeforeAlignBackWard(
				q_begin,
				q.getLengthArray(),
				targetIDArray,
				queryIDArray,
				targetIndexArray,
				queryIndexArray,
				tHitLengthArray,
				qHitLengthArray,
				matchNumArray,
				scoreArray);
		initTempNodeArray<<<initTempNodeBlock, initBlockDim_x>>>(
				hitNum,
				allowableGap,
				raw_pointer_cast( &*tempNodeArray_score     .begin() ),
				raw_pointer_cast( &*tempNodeArray_vertical  .begin() ),
				raw_pointer_cast( &*tempNodeArray_horizontal.begin() ),
				raw_pointer_cast( &*tempNodeArray_matchNum  .begin() )
		);
		localAlignBackward<<<alignNodeBlock, alignBlockDim_x>>>(
				hitNum,
				allowableGap,
				t_begin,
				q_begin,
				raw_pointer_cast( &*h.getTarget().getGateway()    .begin() ),
				raw_pointer_cast( &*h.getTarget().getLengthArray().begin() ),
				raw_pointer_cast( &*h.getTarget().getBaseArray()  .begin() ),
				raw_pointer_cast( &*q.getGateway()    .begin() ),
				raw_pointer_cast( &*q.getLengthArray().begin() ),
				raw_pointer_cast( &*q.getBaseArray()  .begin() ),
				raw_pointer_cast( &*targetIDArray   .begin() ),
				raw_pointer_cast( &*queryIDArray    .begin() ),
				raw_pointer_cast( &*targetIndexArray.begin() ),
				raw_pointer_cast( &*queryIndexArray .begin() ),
				raw_pointer_cast( &*tHitLengthArray .begin() ),
				raw_pointer_cast( &*qHitLengthArray .begin() ),
				raw_pointer_cast( &*matchNumArray   .begin() ),
				raw_pointer_cast( &*scoreArray      .begin() ),
				raw_pointer_cast( &*tempNodeArray_score     .begin() ),
				raw_pointer_cast( &*tempNodeArray_vertical  .begin() ),
				raw_pointer_cast( &*tempNodeArray_horizontal.begin() ),
				raw_pointer_cast( &*tempNodeArray_matchNum  .begin() )
		);
		sortBeforeAlignForward(
				targetIDArray,
				queryIDArray,
				targetIndexArray,
				queryIndexArray,
				tHitLengthArray,
				qHitLengthArray,
				matchNumArray,
				scoreArray);
                initTempNodeArray<<<initTempNodeBlock, initBlockDim_x>>>(
                                hitNum,
                                allowableGap,
                                raw_pointer_cast( &*tempNodeArray_score     .begin() ),
                                raw_pointer_cast( &*tempNodeArray_vertical  .begin() ),
                                raw_pointer_cast( &*tempNodeArray_horizontal.begin() ),
                                raw_pointer_cast( &*tempNodeArray_matchNum  .begin() )
                );
		localAlignForward<<<alignNodeBlock, alignBlockDim_x>>>(
				hitNum,
				allowableGap,
				t_begin,
				q_begin,
				raw_pointer_cast( &*h.getTarget().getGateway()    .begin() ),
				raw_pointer_cast( &*h.getTarget().getLengthArray().begin() ),
				raw_pointer_cast( &*h.getTarget().getBaseArray()  .begin() ),
				raw_pointer_cast( &*q.getGateway()    .begin() ),
				raw_pointer_cast( &*q.getLengthArray().begin() ),
				raw_pointer_cast( &*q.getBaseArray()  .begin() ),
				raw_pointer_cast( &*targetIDArray   .begin() ),
				raw_pointer_cast( &*queryIDArray    .begin() ),
				raw_pointer_cast( &*targetIndexArray.begin() ),
				raw_pointer_cast( &*queryIndexArray .begin() ),
				raw_pointer_cast( &*tHitLengthArray .begin() ),
				raw_pointer_cast( &*qHitLengthArray .begin() ),
				raw_pointer_cast( &*matchNumArray   .begin() ),
				raw_pointer_cast( &*scoreArray      .begin() ),
				raw_pointer_cast( &*tempNodeArray_score     .begin() ),
				raw_pointer_cast( &*tempNodeArray_vertical  .begin() ),
				raw_pointer_cast( &*tempNodeArray_horizontal.begin() ),
				raw_pointer_cast( &*tempNodeArray_matchNum  .begin() )
		);
	} else {
		sortBeforeAlignBackWard(
				q_begin,
				q.getLengthArray(),
				targetIDArray,
				queryIDArray,
				targetIndexArray,
				queryIndexArray,
				tHitLengthArray,
				qHitLengthArray,
				matchNumArray,
				scoreArray);
		initTempNodeArray<<<initTempNodeBlock, initBlockDim_x>>>(
				hitNum,
				allowableGap,
				raw_pointer_cast( &*tempNodeArray_score     .begin() ),
				raw_pointer_cast( &*tempNodeArray_vertical  .begin() ),
				raw_pointer_cast( &*tempNodeArray_horizontal.begin() ),
				raw_pointer_cast( &*tempNodeArray_matchNum  .begin() )
		);
		globalAlignBackward<<<alignNodeBlock, alignBlockDim_x>>>(
				hitNum,
				allowableGap,
				t_begin,
				q_begin,
				raw_pointer_cast( &*h.getTarget().getGateway()    .begin() ),
				raw_pointer_cast( &*h.getTarget().getLengthArray().begin() ),
				raw_pointer_cast( &*h.getTarget().getBaseArray()  .begin() ),
				raw_pointer_cast( &*q.getGateway()    .begin() ),
				raw_pointer_cast( &*q.getLengthArray().begin() ),
				raw_pointer_cast( &*q.getBaseArray()  .begin() ),
				raw_pointer_cast( &*targetIDArray   .begin() ),
				raw_pointer_cast( &*queryIDArray    .begin() ),
				raw_pointer_cast( &*targetIndexArray.begin() ),
				raw_pointer_cast( &*queryIndexArray .begin() ),
				raw_pointer_cast( &*tHitLengthArray .begin() ),
				raw_pointer_cast( &*qHitLengthArray .begin() ),
				raw_pointer_cast( &*matchNumArray   .begin() ),
				raw_pointer_cast( &*scoreArray      .begin() ),
				raw_pointer_cast( &*tempNodeArray_score     .begin() ),
				raw_pointer_cast( &*tempNodeArray_vertical  .begin() ),
				raw_pointer_cast( &*tempNodeArray_horizontal.begin() ),
				raw_pointer_cast( &*tempNodeArray_matchNum  .begin() )
		);
		sortBeforeAlignForward(
				targetIDArray,
				queryIDArray,
				targetIndexArray,
				queryIndexArray,
				tHitLengthArray,
				qHitLengthArray,
				matchNumArray,
				scoreArray);
		initTempNodeArray<<<initTempNodeBlock, initBlockDim_x>>>(
				hitNum,
				allowableGap,
				raw_pointer_cast( &*tempNodeArray_score     .begin() ),
				raw_pointer_cast( &*tempNodeArray_vertical  .begin() ),
				raw_pointer_cast( &*tempNodeArray_horizontal.begin() ),
				raw_pointer_cast( &*tempNodeArray_matchNum  .begin() )
		);
		globalAlignForward<<<alignNodeBlock, alignBlockDim_x>>>(
				hitNum,
				allowableGap,
				t_begin,
				q_begin,
				raw_pointer_cast( &*h.getTarget().getGateway()    .begin() ),
				raw_pointer_cast( &*h.getTarget().getLengthArray().begin() ),
				raw_pointer_cast( &*h.getTarget().getBaseArray()  .begin() ),
				raw_pointer_cast( &*q.getGateway()    .begin() ),
				raw_pointer_cast( &*q.getLengthArray().begin() ),
				raw_pointer_cast( &*q.getBaseArray()  .begin() ),
				raw_pointer_cast( &*targetIDArray   .begin() ),
				raw_pointer_cast( &*queryIDArray    .begin() ),
				raw_pointer_cast( &*targetIndexArray.begin() ),
				raw_pointer_cast( &*queryIndexArray .begin() ),
				raw_pointer_cast( &*tHitLengthArray .begin() ),
				raw_pointer_cast( &*qHitLengthArray .begin() ),
				raw_pointer_cast( &*matchNumArray   .begin() ),
				raw_pointer_cast( &*scoreArray      .begin() ),
				raw_pointer_cast( &*tempNodeArray_score     .begin() ),
				raw_pointer_cast( &*tempNodeArray_vertical  .begin() ),
				raw_pointer_cast( &*tempNodeArray_horizontal.begin() ),
				raw_pointer_cast( &*tempNodeArray_matchNum  .begin() )
		);
	}
        #ifdef TIME_ATTACK
                std::cout << "......................................finished.";
                hipEventRecord( stop, 0 );
                hipEventSynchronize( stop );
                hipEventElapsedTime( &elapsed_time_ms, start, stop );
                std::cout
                                << " (costs " << elapsed_time_ms << "ms) "
                                << targetIDArray.size() << " hits found."
                                << std::endl;
        #endif /* TIME_ATTACK */
	#ifdef MODE_TEST
		CTest::printIsolatedHit(
				targetIDArray,
				targetIndexArray,
				queryIDArray,
				queryIndexArray,
				hitLengthArray);
	#endif /* MODE_TEST */
}
